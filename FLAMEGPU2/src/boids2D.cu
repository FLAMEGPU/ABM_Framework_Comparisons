#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>

#include "flamegpu/flamegpu.h"

/**
 * FLAME GPU 2 implementation of the Boids flocking model in 2D, using spatial2D messaging.
 * This is based on the FLAME GPU 1 implementation, but with dynamic generation of agents. 
 * Agents are also clamped to be within the environment bounds, rather than wrapped as in FLAME GPU 1.
 */

/**
 * Get the length of a vector
 * @param x x component of the vector
 * @param y y component of the vector
 * @return the length of the vector
 */ 
FLAMEGPU_HOST_DEVICE_FUNCTION float vec3Length(const float x, const float y) {
    return sqrtf(x * x + y * y);
}

/**
 * Add a scalar to a vector in-place
 * @param x x component of the vector
 * @param y y component of the vector
 * @param value scalar value to add
 */ 
FLAMEGPU_HOST_DEVICE_FUNCTION void vec3Add(float &x, float &y, const float value) {
    x += value;
    y += value;
}

/**
 * Subtract a scalar from a vector in-place
 * @param x x component of the vector
 * @param y y component of the vector
 * @param value scalar value to subtract
 */ 
FLAMEGPU_HOST_DEVICE_FUNCTION void vec3Sub(float &x, float &y, const float value) {
    x -= value;
    y -= value;
}

/**
 * Multiply a vector by a scalar value in-place
 * @param x x component of the vector
 * @param y y component of the vector
 * @param multiplier scalar value to multiply by
 */ 
FLAMEGPU_HOST_DEVICE_FUNCTION void vec3Mult(float &x, float &y, const float multiplier) {
    x *= multiplier;
    y *= multiplier;
}

/**
 * Divide a vector by a scalar value in-place
 * @param x x component of the vector
 * @param y y component of the vector
 * @param divisor scalar value to divide by
 */ 
FLAMEGPU_HOST_DEVICE_FUNCTION void vec3Div(float &x, float &y, const float divisor) {
    x /= divisor;
    y /= divisor;
}

/**
 * Normalize a 3 component vector in-place
 * @param x x component of the vector
 * @param y y component of the vector
 */ 
FLAMEGPU_HOST_DEVICE_FUNCTION void vec3Normalize(float &x, float &y) {
    // Get the length
    float length = vec3Length(x, y);
    vec3Div(x, y, length);
}

/**
 * Ensure that the x and y position are withini the defined boundary area, wrapping to the far side if out of bounds.
 * Performs the operation in place
 * @param x x component of the vector
 * @param y y component of the vector
 * @param MIN_POSITION the minimum value for each component
 * @param MAX_POSITION the maximum value for each component
 */
FLAMEGPU_HOST_DEVICE_FUNCTION void wrapPosition(float &x, float &y, const float MIN_POSITION, const float MAX_POSITION) {
    const float WIDTH = MAX_POSITION - MIN_POSITION;
    if (x < MIN_POSITION) {
        x += WIDTH;
    }
    if (y < MIN_POSITION) {
        y += WIDTH;
    }
    
    if (x > MAX_POSITION) {
        x -= WIDTH;
    }
    if (y > MAX_POSITION) {
        y -= WIDTH;
    }
}

/**
 * outputdata agent function for Boid agents, which outputs publicly visible properties to a message list
 */
FLAMEGPU_AGENT_FUNCTION(outputdata, flamegpu::MessageNone, flamegpu::MessageSpatial2D) {
    // Output each agents publicly visible properties.
    FLAMEGPU->message_out.setVariable<flamegpu::id_t>("id", FLAMEGPU->getID());
    FLAMEGPU->message_out.setVariable<float>("x", FLAMEGPU->getVariable<float>("x"));
    FLAMEGPU->message_out.setVariable<float>("y", FLAMEGPU->getVariable<float>("y"));
    FLAMEGPU->message_out.setVariable<float>("fx", FLAMEGPU->getVariable<float>("fx"));
    FLAMEGPU->message_out.setVariable<float>("fy", FLAMEGPU->getVariable<float>("fy"));
    return flamegpu::ALIVE;
}

/**
 * inputdata agent function for Boid agents, which reads data from neighbouring Boid agents, to perform the boid flocking model.
 */
FLAMEGPU_AGENT_FUNCTION(inputdata, flamegpu::MessageSpatial2D, flamegpu::MessageNone) {
    // Agent properties in local register
    const flamegpu::id_t id = FLAMEGPU->getID();
    // Agent position
    float agent_x = FLAMEGPU->getVariable<float>("x");
    float agent_y = FLAMEGPU->getVariable<float>("y");
    // Agent velocity
    float agent_fx = FLAMEGPU->getVariable<float>("fx");
    float agent_fy = FLAMEGPU->getVariable<float>("fy");

    // Boids percieved center
    float perceived_centre_x = 0.0f;
    float perceived_centre_y = 0.0f;
    int perceived_count = 0;

    // Boids global velocity matching
    float global_velocity_x = 0.0f;
    float global_velocity_y = 0.0f;

    // Total change in velocity
    float velocity_change_x = 0.f;
    float velocity_change_y = 0.f;

    const float INTERACTION_RADIUS = FLAMEGPU->environment.getProperty<float>("INTERACTION_RADIUS");
    const float SEPARATION_RADIUS = FLAMEGPU->environment.getProperty<float>("SEPARATION_RADIUS");
    // Iterate location messages, accumulating relevant data and counts.
    for (const auto &message : FLAMEGPU->message_in.wrap(agent_x, agent_y)) {
        // Ignore self messages.
        if (message.getVariable<flamegpu::id_t>("id") != id) {
            // Get the message location and velocity.
            const float message_x = message.getVirtualX();
            const float message_y = message.getVirtualY();

            // Check interaction radius
            float separation = vec3Length(agent_x - message_x, agent_y - message_y);

            if (separation < INTERACTION_RADIUS) {
                // Update the percieved centre
                perceived_centre_x += message_x;
                perceived_centre_y += message_y;
                perceived_count++;

                // Update percieved velocity matching
                const float message_fx = message.getVariable<float>("fx");
                const float message_fy = message.getVariable<float>("fy");
                global_velocity_x += message_fx;
                global_velocity_y += message_fy;

                // Update collision centre
                if (separation < (SEPARATION_RADIUS)) {  // dependant on model size
                    // Rule 3) Avoid other nearby boids (Separation)
                    float normalizedSeparation = (separation / SEPARATION_RADIUS);
                    float invNormSep = (1.0f - normalizedSeparation);
                    float invSqSep = invNormSep * invNormSep;

                    const float collisionScale = FLAMEGPU->environment.getProperty<float>("COLLISION_SCALE");
                    velocity_change_x += collisionScale * (agent_x - message_x) * invSqSep;
                    velocity_change_y += collisionScale * (agent_y - message_y) * invSqSep;
                }
            }
        }
    }

    if (perceived_count) {
        // Divide positions/velocities by relevant counts.
        vec3Div(perceived_centre_x, perceived_centre_y, perceived_count);
        vec3Div(global_velocity_x, global_velocity_y, perceived_count);

        // Rule 1) Steer towards perceived centre of flock (Cohesion)
        float steer_velocity_x = 0.f;
        float steer_velocity_y = 0.f;

        const float STEER_SCALE = FLAMEGPU->environment.getProperty<float>("STEER_SCALE");
        steer_velocity_x = (perceived_centre_x - agent_x) * STEER_SCALE;
        steer_velocity_y = (perceived_centre_y - agent_y) * STEER_SCALE;

        velocity_change_x += steer_velocity_x;
        velocity_change_y += steer_velocity_y;

        // Rule 2) Match neighbours speeds (Alignment)
        float match_velocity_x = 0.f;
        float match_velocity_y = 0.f;

        const float MATCH_SCALE = FLAMEGPU->environment.getProperty<float>("MATCH_SCALE");
        match_velocity_x = global_velocity_x * MATCH_SCALE;
        match_velocity_y = global_velocity_y * MATCH_SCALE;

        velocity_change_x += match_velocity_x - agent_fx;
        velocity_change_y += match_velocity_y - agent_fy;
    }

    // Global scale of velocity change
    vec3Mult(velocity_change_x, velocity_change_y, FLAMEGPU->environment.getProperty<float>("GLOBAL_SCALE"));

    // Update agent velocity
    agent_fx += velocity_change_x;
    agent_fy += velocity_change_y;

    // Bound velocity
    float agent_fscale = vec3Length(agent_fx, agent_fy);
    if (agent_fscale > 1) {
        vec3Div(agent_fx, agent_fy, agent_fscale);
    }

    float minSpeed = 0.5f;
    if (agent_fscale < minSpeed) {
        // Normalise
        vec3Div(agent_fx, agent_fy, agent_fscale);

        // Scale to min
        vec3Mult(agent_fx, agent_fy, minSpeed);
    }

    // Apply the velocity
    const float TIME_SCALE = FLAMEGPU->environment.getProperty<float>("TIME_SCALE");
    agent_x += agent_fx * TIME_SCALE;
    agent_y += agent_fy * TIME_SCALE;

    // Wramp position
    const float MIN_POSITION = FLAMEGPU->environment.getProperty<float>("MIN_POSITION");
    const float MAX_POSITION = FLAMEGPU->environment.getProperty<float>("MAX_POSITION");
    wrapPosition(agent_x, agent_y, MIN_POSITION, MAX_POSITION);

    // Update global agent memory.
    FLAMEGPU->setVariable<float>("x", agent_x);
    FLAMEGPU->setVariable<float>("y", agent_y);

    FLAMEGPU->setVariable<float>("fx", agent_fx);
    FLAMEGPU->setVariable<float>("fy", agent_fy);

    return flamegpu::ALIVE;
}

int main(int argc, const char ** argv) {
    flamegpu::ModelDescription model("Boids Spatial3D");

    // Environment variables with default values
    flamegpu::EnvironmentDescription &env = model.Environment();

    // Population size to generate, if no agents are loaded from disk
    env.newProperty("POPULATION_TO_GENERATE", 300u);

    // Environment Bounds
    env.newProperty("MIN_POSITION", -0.5f);
    env.newProperty("MAX_POSITION", +0.5f);

    // Initialisation parameter(s)
    env.newProperty("MAX_INITIAL_SPEED", 1.0f);
    env.newProperty("MIN_INITIAL_SPEED", 0.1f);

    // Interaction radius
    env.newProperty("INTERACTION_RADIUS", 0.05f);
    env.newProperty("SEPARATION_RADIUS", 0.01f);

    // Global Scalers
    env.newProperty("TIME_SCALE", 0.0005f);
    env.newProperty("GLOBAL_SCALE", 0.15f);

    // Rule scalers
    env.newProperty("STEER_SCALE", 0.055f);
    env.newProperty("COLLISION_SCALE", 10.0f);
    env.newProperty("MATCH_SCALE", 0.015f);


    // Define the Location 2D spatial message list
    flamegpu::MessageSpatial2D::Description &message = model.newMessage<flamegpu::MessageSpatial2D>("location");
    // Set the range and bounds.
    message.setRadius(env.getProperty<float>("INTERACTION_RADIUS"));
    message.setMin(env.getProperty<float>("MIN_POSITION"), env.getProperty<float>("MIN_POSITION"));
    message.setMax(env.getProperty<float>("MAX_POSITION"), env.getProperty<float>("MAX_POSITION"));

    // A message to hold the location of an agent.
    message.newVariable<flamegpu::id_t>("id");
    // Spatial 2D messages implicitly have float members x and y, so they do not need to be defined
    message.newVariable<float>("fx");
    message.newVariable<float>("fy");
    message.newVariable<float>("fz");

    // Boid agent
    flamegpu::AgentDescription &agent = model.newAgent("Boid");
    agent.newVariable<float>("x");
    agent.newVariable<float>("y");
    agent.newVariable<float>("fx");
    agent.newVariable<float>("fy");
    // Define the agents methods
    flamegpu::AgentFunctionDescription& outputdataDescription = agent.newFunction("outputdata", outputdata);
    outputdataDescription.setMessageOutput("location");
    flamegpu::AgentFunctionDescription& inputdataDescription = agent.newFunction("inputdata", inputdata);
    inputdataDescription.setMessageInput("location");

    // Specify agent method dependencies, i.e. the exeuction order within a layer.
    model.addExecutionRoot(outputdataDescription);
    inputdataDescription.dependsOn(outputdataDescription);
    // Build the execution graph
    model.generateLayers();

    // Create Model Runner
    flamegpu::CUDASimulation simulator(model);

    // If enabled, define the visualsiation for the model
#ifdef VISUALISATION
    flamegpu::visualiser::ModelVis &visualisation = simulator.getVisualisation();
    {
        flamegpu::EnvironmentDescription &env = model.Environment();
        const float ENV_WIDTH = env.getProperty<float>("MAX_POSITION") - env.getProperty<float>("MIN_POSITION");
        const float ENV_CENTER = env.getProperty<float>("MIN_POSITION") + (ENV_WIDTH) / 2.0f;
        const float INIT_CAM_DISTANCE = 1.05f;
        const float INIT_CAM = env.getProperty<float>("MAX_POSITION") * 1.25f;
        visualisation.setInitialCameraLocation(ENV_CENTER, ENV_CENTER, INIT_CAM_DISTANCE);
        visualisation.setInitialCameraTarget(ENV_CENTER, ENV_CENTER, 0.0f);
        visualisation.setCameraSpeed(0.001f * ENV_WIDTH);
        visualisation.setViewClips(0.00001f, 50);
        auto &circ_agt = visualisation.addAgent("Boid");
        // Position vars are named x, y so they are used by default
        circ_agt.setForwardXVariable("fx");
        circ_agt.setForwardYVariable("fy");
        circ_agt.setModel(flamegpu::visualiser::Stock::Models::STUNTPLANE);
        circ_agt.setModelScale(env.getProperty<float>("SEPARATION_RADIUS")/3.0f);
        // Add a settings UI
        flamegpu::visualiser::PanelVis ui = visualisation.newUIPanel("Environment");
        ui.newStaticLabel("Interaction");
        ui.newEnvironmentPropertyDrag<float>("INTERACTION_RADIUS", 0.0f, 0.05f, 0.001f);
        ui.newEnvironmentPropertyDrag<float>("SEPARATION_RADIUS", 0.0f, 0.05f, 0.001f);
        ui.newStaticLabel("Environment Scalars");
        ui.newEnvironmentPropertyDrag<float>("TIME_SCALE", 0.0f, 1.0f, 0.0001f);
        ui.newEnvironmentPropertyDrag<float>("GLOBAL_SCALE", 0.0f, 0.5f, 0.001f);
        ui.newStaticLabel("Force Scalars");
        ui.newEnvironmentPropertyDrag<float>("STEER_SCALE", 0.0f, 10.0f, 0.001f);
        ui.newEnvironmentPropertyDrag<float>("COLLISION_SCALE", 0.0f, 10.0f, 0.001f);
        ui.newEnvironmentPropertyDrag<float>("MATCH_SCALE", 0.0f, 10.0f, 0.001f);
    }
    visualisation.activate();
#endif

    // Initialisation
    simulator.initialise(argc, argv);

    // If no agent states were provided, generate a population of randomly distributed agents within the environment space
    if (simulator.getSimulationConfig().input_file.empty()) {
        flamegpu::EnvironmentDescription &env = model.Environment();
        // Uniformly distribute agents within space, with uniformly distributed initial velocity.
        // c++ random number generator engine
        std::mt19937_64 rngEngine(simulator.getSimulationConfig().random_seed);
        // Uniform distribution for agent position components
        std::uniform_real_distribution<float> position_distribution(env.getProperty<float>("MIN_POSITION"), env.getProperty<float>("MAX_POSITION"));
        // Uniform distribution of velocity direction components
        std::uniform_real_distribution<float> velocity_distribution(-1, 1);
        // Uniform distribution of velocity magnitudes
        std::uniform_real_distribution<float> velocity_magnitude_distribution(env.getProperty<float>("MIN_INITIAL_SPEED"), env.getProperty<float>("MAX_INITIAL_SPEED"));

        // Generate a population of agents, based on the relevant environment property
        const unsigned int populationSize = env.getProperty<unsigned int>("POPULATION_TO_GENERATE");
        flamegpu::AgentVector population(model.Agent("Boid"), populationSize);
        for (unsigned int i = 0; i < populationSize; i++) {
            flamegpu::AgentVector::Agent instance = population[i];

            // Agent position in space
            instance.setVariable<float>("x", position_distribution(rngEngine));
            instance.setVariable<float>("y", position_distribution(rngEngine));

            // Generate a random velocity direction
            float fx = velocity_distribution(rngEngine);
            float fy = velocity_distribution(rngEngine);
            // Generate a random speed between 0 and the maximum initial speed
            float fmagnitude = velocity_magnitude_distribution(rngEngine);
            // Use the random speed for the velocity.
            vec3Normalize(fx, fy);
            vec3Mult(fx, fy, fmagnitude);

            // Set these for the agent.
            instance.setVariable<float>("fx", fx);
            instance.setVariable<float>("fy", fy);
        }
        simulator.setPopulationData(population);
    }

    // Execute the simulation
    simulator.simulate();

    // Print the exeuction time to stdout
    fprintf(stdout, "Elapsed (s): %.6f\n", simulator.getElapsedTimeSimulation());

    // Join the visualsition if required
#ifdef VISUALISATION
    visualisation.join();
#endif
    return EXIT_SUCCESS;
}
